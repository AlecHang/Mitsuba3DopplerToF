#include "hip/hip_runtime.h"
#include <hip/hip_math_constants.h>

#include <mitsuba/render/optix/matrix.cuh>
#include <mitsuba/render/optix/common.h>

// Include all shapes CUDA headers to generate their PTX programs
#include <mitsuba/render/optix/shapes.h>

extern "C" __global__ void __raygen__rg() {
    unsigned int launch_index = calculate_launch_index();

    // Get inputs from the params struct
    Vector3f ro = Vector3f(params.in_o[0][launch_index],
                           params.in_o[1][launch_index],
                           params.in_o[2][launch_index]),
             rd = Vector3f(params.in_d[0][launch_index],
                           params.in_d[1][launch_index],
                           params.in_d[2][launch_index]);
    float mint = params.in_mint[launch_index],
          maxt = params.in_maxt[launch_index];

    // Replace inf with very large float value as it isn't supported by Optix
    if (maxt == HIP_INF_F)
        maxt = HIP_MAX_NORMAL_F;

    if (params.is_ray_test()) {
        if (!params.in_mask[launch_index]) {
            params.out_hit[launch_index] = false;
        } else {
            optixTrace(
                params.handle,
                make_float3(ro), make_float3(rd),
                mint, maxt, 0.0f,
                OptixVisibilityMask(1),
                OPTIX_RAY_FLAG_TERMINATE_ON_FIRST_HIT,
                0, 1, 0
                );
        }
    } else {
        if (!params.in_mask[launch_index]) {
            params.out_shape_registry_id[launch_index] = 0;
            params.out_t[launch_index] = HIP_INF_F;
        } else {
            // Payload values
            unsigned int payload_t,
                         payload_prim_u,
                         payload_prim_v,
                         payload_prim_index,
                         payload_shape_ptr,
                         payload_inst_index;
            // Initialize shape pointer to null
            payload_shape_ptr = 0u;
            // Instance index is initialized to 0 when there is no instancing in the scene
            payload_inst_index = params.out_inst_index[launch_index];

            optixTrace(
                params.handle,
                make_float3(ro), make_float3(rd),
                mint, maxt, 0.0f,
                OptixVisibilityMask(1),
                OPTIX_RAY_FLAG_NONE,
                0, 1, 0,
                payload_t,
                payload_prim_u,
                payload_prim_v,
                payload_prim_index,
                payload_shape_ptr,
                payload_inst_index
            );

            // Write preliminary intersection to output data pointers
            params.out_t[launch_index]          = __int_as_float(payload_t);
            params.out_prim_uv[0][launch_index] = __int_as_float(payload_prim_u);
            params.out_prim_uv[1][launch_index] = __int_as_float(payload_prim_v);
            params.out_prim_index[launch_index] = payload_prim_index;
            params.out_shape_registry_id[launch_index]  = payload_shape_ptr;
            params.out_inst_index[launch_index] = payload_inst_index;
        }
    }
}

extern "C" __global__ void __miss__ms() {
    unsigned int launch_index = calculate_launch_index();

    if (params.is_ray_test()) {
        params.out_hit[launch_index] = false;
    } else {
        optixSetPayload_0(__float_as_int(HIP_INF_F));
        optixSetPayload_4(0);
    }
}

struct OptixException {
    int code;
    const char* string;
};

__constant__ OptixException exceptions[] = {
    { OPTIX_EXCEPTION_CODE_STACK_OVERFLOW, "OPTIX_EXCEPTION_CODE_STACK_OVERFLOW" },
    { OPTIX_EXCEPTION_CODE_TRACE_DEPTH_EXCEEDED, "OPTIX_EXCEPTION_CODE_TRACE_DEPTH_EXCEEDED" },
    { OPTIX_EXCEPTION_CODE_TRAVERSAL_DEPTH_EXCEEDED, "OPTIX_EXCEPTION_CODE_TRAVERSAL_DEPTH_EXCEEDED" },
    { OPTIX_EXCEPTION_CODE_TRAVERSAL_INVALID_TRAVERSABLE, "OPTIX_EXCEPTION_CODE_TRAVERSAL_INVALID_TRAVERSABLE" },
    { OPTIX_EXCEPTION_CODE_TRAVERSAL_INVALID_MISS_SBT, "OPTIX_EXCEPTION_CODE_TRAVERSAL_INVALID_MISS_SBT" },
    { OPTIX_EXCEPTION_CODE_TRAVERSAL_INVALID_HIT_SBT, "OPTIX_EXCEPTION_CODE_TRAVERSAL_INVALID_HIT_SBT" }
};

extern "C" __global__ void __exception__err() {
    int ex_code = optixGetExceptionCode();
    printf("Optix Exception %u: %s\n", ex_code, exceptions[ex_code].string);
    // TODO: retreive more informations based on exception
}
